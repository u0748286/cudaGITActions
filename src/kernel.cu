#include "hip/hip_runtime.h"

#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <cmath>

const double M_PI = 3.1415926;
const char* vertexShaderSource = R"(
#version 330 core
layout(location = 0) in vec3 aPos;  // 顶点位置
layout(location = 1) in vec3 aColor; // 顶点颜色
out vec3 vertexColor; // 从顶点着色器传递到片段着色器的颜色
uniform float angle;  // 旋转角度

void main()
{
    // 旋转矩阵
    mat4 rotation = mat4(
        cos(angle), -sin(angle), 0.0, 0.0,
        sin(angle), cos(angle), 0.0, 0.0,
        0.0, 0.0, 1.0, 0.0,
        0.0, 0.0, 0.0, 1.0
    );

    gl_Position = rotation * vec4(aPos, 1.0); // 应用旋转
    vertexColor = aColor; // 传递颜色
}
)";

const char* fragmentShaderSource = R"(
#version 330 core
in vec3 vertexColor; // 接收来自顶点着色器的颜色
out vec4 fragColor;  // 输出的片段颜色

void main()
{
    fragColor = vec4(vertexColor, 1.0); // 设置片段颜色
}
)";

float angle = 0.0f; // 初始化旋转角度

int main() {
    // 初始化 GLFW
    if (!glfwInit()) {
        return -1;
    }

    // 创建 OpenGL 3.3 上下文窗口
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow* window = glfwCreateWindow(800, 600, "OpenGL Rotating Quad", NULL, NULL);
    if (!window) {
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);
    glewInit();

    // 定义顶点数据 (两个三角形组成的四边形)
    float vertices[] = {
        // 位置           // 颜色
        -0.5f, -0.5f, 0.0f,  1.0f, 0.0f, 0.0f, // 左下角 (红色)
         0.5f, -0.5f, 0.0f,  0.0f, 1.0f, 0.0f, // 右下角 (绿色)
         0.5f,  0.5f, 0.0f,  0.0f, 0.0f, 1.0f, // 右上角 (蓝色)
         -0.5f,  0.5f, 0.0f,  1.0f, 1.0f, 0.0f  // 左上角 (黄色)
    };

    unsigned int indices[] = {
        0, 1, 2, // 第一个三角形
        0, 2, 3  // 第二个三角形
    };

    unsigned int VBO, VAO, EBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    // 绑定 VAO
    glBindVertexArray(VAO);

    // 绑定 VBO
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    // 绑定 EBO
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

    // 设置顶点属性指针
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)(3 * sizeof(float)));
    glEnableVertexAttribArray(1);

    // 编译和链接着色器程序
    GLuint vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);

    GLuint fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);

    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);

    // 删除着色器，因为它们现在已经链接到程序中
    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);

    // 主循环
    while (!glfwWindowShouldClose(window)) {
        // 计算旋转角度
        angle += 0.01f; // 每帧增加旋转角度
        if (angle > 2 * M_PI) angle -= 2 * M_PI; // 确保角度在0到2π之间

        // 清除屏幕
        glClear(GL_COLOR_BUFFER_BIT);

        // 使用着色器程序
        glUseProgram(shaderProgram);

        // 设置旋转角度的 uniform 变量
        glUniform1f(glGetUniformLocation(shaderProgram, "angle"), angle);

        // 绘制四边形 (两个三角形)
        glBindVertexArray(VAO);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

        // 交换缓冲区和轮询事件
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // 清理资源
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);
    glfwTerminate();
    return 0;
}
